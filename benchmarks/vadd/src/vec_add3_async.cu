#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstdlib>

#define MAX_STREAMS 6
#define NUM_TILES 1
#define SHIFTS 24
#define K_LOOP 1

#define CHECK_CUDA(call)                                              \
    do {                                                              \
        hipError_t err = call;                                       \
        if (err != hipSuccess) {                                     \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)    \
                      << " at " << __FILE__ << ":" << __LINE__        \
                      << std::endl;                                   \
            exit(EXIT_FAILURE);                                       \
        }                                                             \
    } while (0)

__global__ void vaddLX(const float* A, const float* B, const float* D, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        for (int k = 0; k < K_LOOP; ++k) {
            C[idx] += A[idx] + B[idx] + D[idx];
            C[idx] = fmodf(C[idx], 256.0f);
        }
    }
}

void cudaWarmUp() {
    const size_t warmup_size = 1024;
    float* h_buf;
    float* d_buf;

    CHECK_CUDA(hipHostMalloc(&h_buf, warmup_size, hipHostMallocDefault));
    CHECK_CUDA(hipMalloc(&d_buf, warmup_size));

    for (size_t i = 0; i < warmup_size / sizeof(float); ++i) {
        h_buf[i] = static_cast<float>(i);
    }

    CHECK_CUDA(hipMemcpy(d_buf, h_buf, warmup_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(h_buf, d_buf, warmup_size, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_buf));
    CHECK_CUDA(hipHostFree(h_buf));
    CHECK_CUDA(hipDeviceSynchronize());
}

void compute_golden(const float* A, const float* B, const float* D, float* C, int N) {
    for (int i = 0; i < N; ++i) {
        for (int k = 0; k < K_LOOP; ++k) {
            C[i] += A[i] + B[i] + D[i];
            C[i] = fmodf(C[i], 256.0f);
        }
    }
}

bool verify_result(const float* C_host, const float* C_device, int N, float epsilon = 1e-5) {
    int mismatches = 0;
    for (int i = 0; i < N; ++i) {
        if (std::fabs(C_host[i] - C_device[i]) > epsilon) {
            mismatches++;
        }
    }
    std::cout << "Number of mismatches: " << mismatches << std::endl;
    return mismatches == 0;
}

int main() {
    const int N = 1 << SHIFTS;
    const size_t size = N * sizeof(float);

    bool verify = false;
    if (const char* env = std::getenv("VERIFY")) {
        verify = std::atoi(env) != 0;
    }

    cudaWarmUp();

    // Allocate host memory
    float *h_A, *h_B, *h_D, *h_C_device, *h_C_golden = nullptr;
    CHECK_CUDA(hipHostMalloc(&h_A, size, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_B, size, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_D, size, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_C_device, size, hipHostMallocDefault));
    if (verify) {
        CHECK_CUDA(hipHostMalloc(&h_C_golden, size, hipHostMallocDefault));
    }

    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i % 256);
        h_B[i] = static_cast<float>((2 * i) % 256);
        h_D[i] = static_cast<float>((3 * i) % 256);
    }

    const int TILE_SIZE = (N + NUM_TILES - 1) / NUM_TILES;
    const size_t TILE_BYTES = TILE_SIZE * sizeof(float);

    float *d_A[NUM_TILES], *d_B[NUM_TILES], *d_D[NUM_TILES], *d_C[NUM_TILES];
    for (int i = 0; i < NUM_TILES; ++i) {
        CHECK_CUDA(hipMalloc(&d_A[i], TILE_BYTES));
        CHECK_CUDA(hipMalloc(&d_B[i], TILE_BYTES));
        CHECK_CUDA(hipMalloc(&d_D[i], TILE_BYTES));
        CHECK_CUDA(hipMalloc(&d_C[i], TILE_BYTES));
    }

    hipStream_t streams[MAX_STREAMS];
    for (int i = 0; i < MAX_STREAMS; ++i) {
        CHECK_CUDA(hipStreamCreate(&streams[i]));
    }

    auto offload_start = std::chrono::high_resolution_clock::now();

    for (int t = 0; t < NUM_TILES; ++t) {
        int offset = t * TILE_SIZE;
        int current_tile_size = std::min(TILE_SIZE, N - offset);
        size_t current_tile_bytes = current_tile_size * sizeof(float);
        int stream_id = t % MAX_STREAMS;

        CHECK_CUDA(hipMemcpyAsync(d_A[t], h_A + offset, current_tile_bytes, hipMemcpyHostToDevice, streams[stream_id]));
        CHECK_CUDA(hipMemcpyAsync(d_B[t], h_B + offset, current_tile_bytes, hipMemcpyHostToDevice, streams[stream_id]));
        CHECK_CUDA(hipMemcpyAsync(d_D[t], h_D + offset, current_tile_bytes, hipMemcpyHostToDevice, streams[stream_id]));

        dim3 threadsPerBlock(256);
        dim3 blocksPerGrid((current_tile_size + threadsPerBlock.x - 1) / threadsPerBlock.x);
        vaddLX<<<blocksPerGrid, threadsPerBlock, 0, streams[stream_id]>>>(
            d_A[t], d_B[t], d_D[t], d_C[t], current_tile_size);

        CHECK_CUDA(hipMemcpyAsync(h_C_device + offset, d_C[t], current_tile_bytes, hipMemcpyDeviceToHost, streams[stream_id]));
    }

    CHECK_CUDA(hipDeviceSynchronize());
    auto offload_end = std::chrono::high_resolution_clock::now();

    if (verify) {
        compute_golden(h_A, h_B, h_D, h_C_golden, N);
        verify_result(h_C_golden, h_C_device, N);
    }

    for (int i = 0; i < NUM_TILES; ++i) {
        CHECK_CUDA(hipFree(d_A[i]));
        CHECK_CUDA(hipFree(d_B[i]));
        CHECK_CUDA(hipFree(d_D[i]));
        CHECK_CUDA(hipFree(d_C[i]));
    }

    for (int i = 0; i < MAX_STREAMS; ++i) {
        CHECK_CUDA(hipStreamDestroy(streams[i]));
    }

    CHECK_CUDA(hipHostFree(h_A));
    CHECK_CUDA(hipHostFree(h_B));
    CHECK_CUDA(hipHostFree(h_D));
    CHECK_CUDA(hipHostFree(h_C_device));
    if (verify) {
        CHECK_CUDA(hipHostFree(h_C_golden));
    }

    std::chrono::duration<double, std::milli> offload_elapsed = offload_end - offload_start;
    std::cout << "Execution Time: " << offload_elapsed.count() << " ms\n";

    return 0;
}
