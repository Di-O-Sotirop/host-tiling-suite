#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <chrono>

#define SHIFTS 16
#define MAX_STREAMS 1
#define NUM_TILES 1
#define TILE_BUFFERS 1
#define GPU_DEVICE 0
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05
// #define VERIFY 1
#define THREADS_PER_BLOCK 256


#ifdef VERIFY
  #define RUN_ON_CPU
#endif

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void warmup_kernel(float *data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 256)
        data[i] += 1.0f;
}

void warmup_cuda_runtime() {
    const int size = 256;
    float *h_buf, *d_buf;
    hipStream_t stream;

    CHECK_CUDA(hipHostMalloc(&h_buf, size * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipMalloc(&d_buf, size * sizeof(float)));
    CHECK_CUDA(hipStreamCreate(&stream));

    for (int i = 0; i < size; ++i)
        h_buf[i] = static_cast<float>(i);

    CHECK_CUDA(hipMemcpyAsync(d_buf, h_buf, size * sizeof(float), hipMemcpyHostToDevice, stream));
    warmup_kernel<<<(size + 127) / 128, 128, 0, stream>>>(d_buf);
    CHECK_CUDA(hipMemcpyAsync(h_buf, d_buf, size * sizeof(float), hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    CHECK_CUDA(hipFree(d_buf));
    CHECK_CUDA(hipHostFree(h_buf));
    CHECK_CUDA(hipStreamDestroy(stream));
}

void init(int n, float *alpha, float *beta, float *A, float *u1, float *v1, float *u2, float *v2, float *w, float *x, float *y, float *z) {
    *alpha = 1.0f;
    *beta = 1.0f;
    for (int i = 0; i < n; ++i) {
        u1[i] = static_cast<float>(i % 64) / n;
        u2[i] = static_cast<float>(i % 64) / (2.0f * n);
        v1[i] = static_cast<float>((i % 64) + 1) / (4.0f * n);
        v2[i] = static_cast<float>((i % 64) + 1) / (1.5f * n);
        y[i] = static_cast<float>((i % 64) + 1) / (3.0f * n);
        z[i] = static_cast<float>((i % 64) + 1) / (5.0f * n);
        x[i] = 0.0f;
        w[i] = 0.0f;

        for (int j = 0; j < n; ++j)
            A[i * n + j] = static_cast<float>((i % 64) * (j % 64));
    }
}

void compareResults(int n, const float *w_cpu, const float *w_gpu) {
    int fail = 0;
    for (int i = 0; i < n; ++i) {
        float diff = std::abs(w_cpu[i] - w_gpu[i]) / std::abs(w_cpu[i]);
        if (diff > PERCENT_DIFF_ERROR_THRESHOLD / 100.0f) {
            fail++;
            std::printf("%f ~ %f\n", w_cpu[i], w_gpu[i]);
        }
    }
    std::printf("Number of mismatches: %d\n", fail);
}

void gemver_cpu(int n, float alpha, float beta, float *A, float *u1, float *v1, float *u2, float *v2, float *w, float *x, float *y, float *z) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            A[i * n + j] += static_cast<int>(u1[i] * v1[j] + u2[i] * v2[j]);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            x[i] += static_cast<int>(beta * A[j * n + i] * y[j]);
        x[i] += z[i];
    }

    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            w[i] += static_cast<int>(alpha * A[i * n + j] * x[j]);
}

__global__ void gemver_kernel1(int n, float *a, const float *v1, const float *v2, const float *u1, const float *u2) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < n) && (j < n))
        a[i * n + j] += u1[i] * v1[j] + u2[i] * v2[j];
}

__global__ void gemver_kernel2(int n, float beta, const float *a, float *x, const float *y, const float *z) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        for (int j = 0; j < n; ++j)
            x[i] += static_cast<int>(beta * a[j * n + i] * y[j]);
        x[i] += z[i];
    }
}

__global__ void gemver_kernel3(int n, float alpha, const float *a, const float *x, float *w) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        for (int j = 0; j < n; ++j)
            w[i] += static_cast<int>(alpha * a[i * n + j] * x[j]);
            // w[j] += static_cast<int>(alpha * a[j * n + i] * x[i]); //Does this alteration give correct results?
    }
}

void gemverCuda(int n, float alpha, float beta,
                float *A, float *u1, float *v1, float *u2, float *v2,
                float *w, float *w_gpu, float *x, float *y, float *z) {

    hipStream_t stream;
    float *A_gpu, *x_gpu, *y_gpu, *z_gpu, *v1_gpu, *v2_gpu, *u1_gpu, *u2_gpu, *w_gpu_d;

    dim3 block1(THREADS_PER_BLOCK,1);
    dim3 grid1((n + block1.x - 1) / block1.x, (n + block1.y - 1) / block1.y);

    dim3 blockX(THREADS_PER_BLOCK);
    dim3 gridX((n + blockX.x - 1) / blockX.x);

    CHECK_CUDA(hipMalloc(&A_gpu, sizeof(float) * n * n));
    CHECK_CUDA(hipMalloc(&x_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&y_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&z_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&w_gpu_d, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&v1_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&v2_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&u1_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMalloc(&u2_gpu, sizeof(float) * n));

    CHECK_CUDA(hipStreamCreate(&stream));


    CHECK_CUDA(hipMemcpyAsync(y_gpu, y, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(z_gpu, z, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(v1_gpu, v1, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(v2_gpu, v2, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(u1_gpu, u1, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(u2_gpu, u2, sizeof(float) * n, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemsetAsync(x_gpu, 0, sizeof(float) * n, stream));
    CHECK_CUDA(hipMemsetAsync(w_gpu_d, 0, sizeof(float) * n, stream));

    auto start = std::chrono::high_resolution_clock::now();
    CHECK_CUDA(hipMemcpyAsync(A_gpu, A, sizeof(float) * n * n, hipMemcpyHostToDevice, stream));
    gemver_kernel1<<<grid1, block1, 0, stream>>>(n, A_gpu, v1_gpu, v2_gpu, u1_gpu, u2_gpu);
     CHECK_CUDA(hipStreamSynchronize(stream));
    auto end = std::chrono::high_resolution_clock::now();
    gemver_kernel2<<<gridX, blockX, 0, stream>>>(n, beta, A_gpu, x_gpu, y_gpu, z_gpu);
    gemver_kernel3<<<gridX, blockX, 0, stream>>>(n, alpha, A_gpu, x_gpu, w_gpu_d);

    CHECK_CUDA(hipMemcpyAsync(w_gpu, w_gpu_d, sizeof(float) * n, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

  
    std::chrono::duration<double, std::milli> elapsed = end - start;
    std::cout << "Execution Time: " << elapsed.count() << " ms\n";

    CHECK_CUDA(hipFree(A_gpu));
    CHECK_CUDA(hipFree(x_gpu));
    CHECK_CUDA(hipFree(y_gpu));
    CHECK_CUDA(hipFree(z_gpu));
    CHECK_CUDA(hipFree(w_gpu_d));
    CHECK_CUDA(hipFree(v1_gpu));
    CHECK_CUDA(hipFree(v2_gpu));
    CHECK_CUDA(hipFree(u1_gpu));
    CHECK_CUDA(hipFree(u2_gpu));
    CHECK_CUDA(hipStreamDestroy(stream));
}

int main() {
    warmup_cuda_runtime();
    auto start = std::chrono::high_resolution_clock::now();
    const int N = 32 * SHIFTS;

    float alpha, beta;
    float *A;
    CHECK_CUDA(hipHostMalloc(&A, N * N * sizeof(float), hipHostMallocDefault));

    float *u1 = (float *)malloc(N * sizeof(float));
    float *v1 = (float *)malloc(N * sizeof(float));
    float *u2 = (float *)malloc(N * sizeof(float));
    float *v2 = (float *)malloc(N * sizeof(float));
    float *w = (float *)malloc(N * sizeof(float));
    float *w_gpu = (float *)malloc(N * sizeof(float));
    float *x = (float *)malloc(N * sizeof(float));
    float *y = (float *)malloc(N * sizeof(float));
    float *z = (float *)malloc(N * sizeof(float));

    init(N, &alpha, &beta, A, u1, v1, u2, v2, w, x, y, z);

    gemverCuda(N, alpha, beta, A, u1, v1, u2, v2, w, w_gpu, x, y, z);

#ifdef RUN_ON_CPU
    gemver_cpu(N, alpha, beta, A, u1, v1, u2, v2, w, x, y, z);
    compareResults(N, w, w_gpu);
#else
    // for (int i = 0; i < N; ++i)
    //     std::cerr << w_gpu[i] << " ";
    // std::cerr << std::endl;
#endif

    CHECK_CUDA(hipHostFree(A));
    free(u1); free(v1); free(u2); free(v2);
    free(w); free(w_gpu); free(x); free(y); free(z);
    auto end = std::chrono::high_resolution_clock::now();
    #ifndef RUN_ON_CPU
        std::chrono::duration<double, std::milli> elapsed = end - start;
        std::cout << "Total Execution Time: " << elapsed.count() << " ms\n";
    #endif
    return 0;
}
